#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hip/hip_fp16.h>

#include "cudaUtil.cuh"

namespace cuUtil {

template<typename T>
__global__ void convertDataType(const size_t n, const float *in, T *out) {
    const size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<T>(in[idx]);
//        printf("in[%d] = %f, static_cast<float>out[%d] = %f\n", idx, in[idx], idx, static_cast<float>(out[idx]));
    }
}

template __global__ void convertDataType<int>(const size_t n, const float *in, int *out);

template __global__ void convertDataType<float>(const size_t n, const float *in, float *out);

template __global__ void convertDataType<double>(const size_t n, const float *in, double *out);

template __global__ void convertDataType<half>(const size_t n, const float *in, half *out);

void makeData(float *data, const int size) {
    // using cuRAND to initialize

    hiprandGenerator_t curandGen;

    hiprandCreateGenerator(&curandGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curandGen, 1337ULL);

    hiprandGenerateUniform(curandGen, data, size);

    hiprandDestroyGenerator(curandGen);
}


}